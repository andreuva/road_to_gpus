/* 
 * purpose:      CUDA managed unified memory for >= pascal architectures;
 *               this version just uses cudaMallocManaged() on the host,
 *               then runs kernels on the GPU to add together two arrays
 *               of size 1 GB and save the results into a third array;
 *               n.b. here we want to again separate the initialization 
 *                    stage but then run the actual calculation in a
 *                    loop over 100 iterations to get a little better
 *                    statistics from the profiler
 * result:       from profiling via 'nvprof ./a.out' we now see greatly
 *               improved compute performance and much better memory
 *               bandwidth at again almost identical page fault statistics
 * compilation:  nvcc ./unified_memory_example_3.cu
 * usage:        ./a.out
 */




#include <hip/hip_runtime.h>
#include <stdio.h>
#define ARRAYDIM 268435456



/* 
 * GPU kernel doing the initialization
 */
__global__ void KrnlDmmyInit(float *x, float *y, float *z)
{
    int i;

    i = (blockIdx.x * blockDim.x) + threadIdx.x;
    x[i] = (float) i;
    y[i] = (float) (i + 1);

    return;
}



/* 
 * GPU kernel doing the calculation, ie adding together two arrays
 */
__global__ void KrnlDmmyCalc(float *x, float *y, float *z)
{
    int i;

    i = (blockIdx.x * blockDim.x) + threadIdx.x;
    z[i] = x[i] + y[i];

    return;
}







/* 
 * host main  
 */
int main()
{
    int i, cudaRtrn;
    dim3 thrds_per_block, blcks_per_grid;
    float *a, *b, *c;

   /* 
    * Let us make use of cudaMallocManaged() to allocate 3 arrays 
    * of size 1 GB each for subsequent usage on the GPU. 
    */
    if (cudaRtrn = hipMallocManaged(&a, ARRAYDIM * sizeof(float)) != 0) {
       printf("*** allocation failed for array a[], %d ***\n", cudaRtrn);
    }
    if (cudaRtrn = hipMallocManaged(&b, ARRAYDIM * sizeof(float)) != 0) {
       printf("*** allocation failed for array b[], %d ***\n", cudaRtrn);
    }
    if (cudaRtrn = hipMallocManaged(&c, ARRAYDIM * sizeof(float)) != 0) {
       printf("*** allocation failed for array c[], %d ***\n", cudaRtrn);
    }

   /* 
    * next we want to call simple kernels that (i) initialize array 
    * elements a[] and b[] with thread-specific values and (ii) add 
    * together these values and store back the results into array c[]
    * the latter within a loop over 100 iterations
    */
    thrds_per_block.x = 256;
    blcks_per_grid.x = ARRAYDIM / thrds_per_block.x;
    KrnlDmmyInit<<<blcks_per_grid, thrds_per_block>>>(a, b, c);
    hipDeviceSynchronize(); 
    for (i=0; i<100; i++) {
        KrnlDmmyCalc<<<blcks_per_grid, thrds_per_block>>>(a, b, c);
        hipDeviceSynchronize();
    }
    hipFree(c);
    hipFree(b);
    hipFree(a);


    return(0);
}
