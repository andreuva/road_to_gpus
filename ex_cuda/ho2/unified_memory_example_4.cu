#include "hip/hip_runtime.h"
/* 
 * purpose:      CUDA managed unified memory for >= pascal architectures;
 *               this version just uses hipMallocManaged() on the host,
 *               then runs kernels on the GPU to add together two arrays
 *               of size 1 GB and save the results into a third array;
 *               n.b. here we want to stick to a separated initialization 
 *                    kernel, but then before running the actual compute
 *                    kernel do the unified memory prefetching and see
 *                    whether this will affect compute/memory bandwith/page
 *                    faults performance;
 * result:       from profiling via 'nvprof ./a.out' we now see pretty
 *               much the best results so far, hence prefetching seems to
 *               really pay off ! interestingly the number of page faults
 *               has also decreased;
 * compilation:  nvcc ./unified_memory_example_4.cu
 * usage:        ./a.out
 */



#include <stdio.h>
#define ARRAYDIM 268435456



/* 
 * GPU kernel doing the initialization
 */
__global__ void KrnlDmmyInit(float *x, float *y, float *z)
{
    int i;

    i = (blockIdx.x * blockDim.x) + threadIdx.x;
    x[i] = (float) i;
    y[i] = (float) (i + 1);

    return;
}



/* 
 * GPU kernel doing the calculation, ie adding together two arrays
 */
__global__ void KrnlDmmyCalc(float *x, float *y, float *z)
{
    int i;

    i = (blockIdx.x * blockDim.x) + threadIdx.x;
    z[i] = x[i] + y[i];

    return;
}







/* 
 * host main  
 */
int main()
{
    int i, cudaRtrn;
    dim3 thrds_per_block, blcks_per_grid;
    float *a, *b, *c;

   /* 
    * Let us make use of hipMallocManaged() to allocate 3 arrays 
    * of size 1 GB each for subsequent usage on the GPU. 
    */
    if (cudaRtrn = hipMallocManaged(&a, ARRAYDIM * sizeof(float)) != 0) {
       printf("*** allocation failed for array a[], %d ***\n", cudaRtrn);
    }
    if (cudaRtrn = hipMallocManaged(&b, ARRAYDIM * sizeof(float)) != 0) {
       printf("*** allocation failed for array b[], %d ***\n", cudaRtrn);
    }
    if (cudaRtrn = hipMallocManaged(&c, ARRAYDIM * sizeof(float)) != 0) {
       printf("*** allocation failed for array c[], %d ***\n", cudaRtrn);
    }

   /* 
    * next we want to call simple kernels that (i) initialize array 
    * elements a[] and b[] with thread-specific values and (ii) add 
    * together these values and store back the results into array c[]
    * where the latter task shall be repeated within a loop over
    * 100 iterations and memory be explicitly sent to the device
    * with the help of prefetching
    */
    thrds_per_block.x = 256;
    blcks_per_grid.x = ARRAYDIM / thrds_per_block.x;
    KrnlDmmyInit<<<blcks_per_grid, thrds_per_block>>>(a, b, c);
    hipDeviceSynchronize(); 
    hipMemPrefetchAsync(a, ARRAYDIM * sizeof(float), 0, NULL);
    hipMemPrefetchAsync(b, ARRAYDIM * sizeof(float), 0, NULL);
    hipMemPrefetchAsync(c, ARRAYDIM * sizeof(float), 0, NULL);
    for (i=0; i<100; i++) {
        KrnlDmmyCalc<<<blcks_per_grid, thrds_per_block>>>(a, b, c);
        hipDeviceSynchronize();
    }
    hipFree(c);
    hipFree(b);
    hipFree(a);


    return(0);
}
