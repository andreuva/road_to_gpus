/*
 *  usage:  nvcc ./stream_test_v2.cu -o ./stream_v2_legacy
 *          nvvp ./stream_v2_legacy    ( or as root: 
 *                                       nvvp -vm /usr/lib64/jvm/jre-1.8.0/bin/java ./stream_v2_legacy )
 *          ... versus ...
 *          nvcc --default-stream per-thread ./stream_test_v2.cu -o ./stream_v2_per-thread
 *          nvvp ./stream_v2_per-thread    ( or as root: 
 *                                           nvvp -vm /usr/lib64/jvm/jre-1.8.0/bin/java ./stream_v2_per-thread )
 *
 *  purpose: just test whether substitution of cudaMalloc() with cudaMallocManaged() 
 *           will work
 *  result:  yes it does, however, one needs to be careful with synchronizing individual
 *           streams before accessing managed memory, hence the below inserted call was
 *           crucial, because without it only default stream 0 had printed out correct
 *           results and all other streams just 0 !
 *           n.b. out-commented line 'cudaMalloc(&data[i]...' would have worked together with 
 *                also commented lines 'cudaMemcpy(... printf(...' in terms of low level 
 *                checking of results
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>


const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x;
    for (int i = tid; i < n; i += blockDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int main()
{
    const int num_streams = 8;
    float localx[N];

    hipStream_t streams[num_streams];
    float *data[num_streams];

    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
 
        //cudaMalloc(&data[i], N * sizeof(float));
        hipMallocManaged(&data[i], N * sizeof(float));
        
        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);
    }

    // and a quick check of results because individual streams 
    // should have done identical calculations !
    for (int i = 0; i < num_streams; i++) {
        // cudaMemcpy(localx, data[i], N * sizeof(float), cudaMemcpyDeviceToHost);
        // printf("*** %d %12.6lf%12.6lf%12.6lf\n", i, localx[0], localx[1], localx[2]);
        hipStreamSynchronize(streams[i]);
        printf("*** %d %12.6lf%12.6lf%12.6lf\n", i, data[i][0], data[i][1], data[i][2]);
    }

    hipDeviceReset();

    return 0;
}

