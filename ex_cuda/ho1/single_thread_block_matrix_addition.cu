/* 
 * purpose:      just a demo to show how matrix addition can be done on 
 *               the GPU with just a single thread block, ie for rather 
 *               small sized underlying matrix dimensions
 * compilation:  nvcc ./single_thread_block_matrix_addition.cu
 * usage:        ./a.out
 */ 


#include <hip/hip_runtime.h>
#include <stdio.h>

#define   N 30

/* 
 * GPU kernel 
 */
__global__ void MatAdd(float **A, float **B, float **C)
{
    int i, j;

    i = threadIdx.x;
    j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}




/* 
 * host main  
 */
int main()
{
    int i, j;
    dim3 threadsPerBlock, numBlocks;
    float **A, **B, **C;

   /* 
    * using CUDA unified memory, first allocate 
    * the memory in convenient 2D format, then 
    * initialize with some dummy content        
    */
    hipMallocManaged(&A, N * sizeof(float *));
    hipMallocManaged(&B, N * sizeof(float *));
    hipMallocManaged(&C, N * sizeof(float *));
    for (i = 0; i < N; i++) {
        hipMallocManaged(&A[i], N * sizeof(float));
        hipMallocManaged(&B[i], N * sizeof(float));
        hipMallocManaged(&C[i], N * sizeof(float));
        for (j = 0; j < N; j++) {
            A[i][j] = (float) ((i * N) + j);
            B[i][j] = (N * N) - A[i][j];
            C[i][j] = (float) 0;
        }
    }

   /* set up GPU kernel execution configuration */
    threadsPerBlock.x = N;
    threadsPerBlock.y = N;
    numBlocks.x = 1;
   
   /* launch the GPU kernel */
    MatAdd<<<numBlocks, threadsPerBlock>>>(A, B, C);  
    hipDeviceSynchronize();
 
   /* print result */
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            printf("%d %d %f\n", i, j, C[i][j]);
        }
    }
   
   /* make clean */
    for (i = 0; i < N; i++) {
        hipFree(C[i]);
        hipFree(B[i]);
        hipFree(A[i]);
    }
    hipFree(C);
    hipFree(B);
    hipFree(A);


    return(0);
}
