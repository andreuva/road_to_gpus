
#include <hip/hip_runtime.h>
/* 
 * GPU kernel 
 */
__global__ void VecAdd(float *A, float *B, float *C)
{
    int i;

    i = threadIdx.x;
    C[i] = A[i] + B[i];
}



extern "C" void ntmdtr_(float *A, float *B, float *C, int *N)
{
    dim3 numBlocks, threadsPerBlock;
    float *AD, *BD, *CD;
    
   /* 
    * set up GPU kernel execution configuration 
    */
    threadsPerBlock.x = *N;
    numBlocks.x = 1;

   /* 
    * prepare device memory as we need to go the explicit
    * cudaMemcpy() way this time
    */
    hipMalloc((void **) &AD, (*N) * sizeof(float));
    hipMalloc((void **) &BD, (*N) * sizeof(float));
    hipMalloc((void **) &CD, (*N) * sizeof(float));
    
    // transfer data to GPU 
    hipMemcpy(AD, A, (*N) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(BD, B, (*N) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(CD, C, (*N) * sizeof(float), hipMemcpyHostToDevice);

    // launch the GPU kernel 
    VecAdd<<<numBlocks, threadsPerBlock>>>(AD, BD, CD);  
    hipDeviceSynchronize();

    // copy back the result from the GPU, A and B should be unchanged !
    hipMemcpy(C, CD, (*N) * sizeof(float), hipMemcpyDeviceToHost);

    // free memory on the GPU
    hipFree(AD);  
    hipFree(BD);  
    hipFree(CD);  

    return;
}
