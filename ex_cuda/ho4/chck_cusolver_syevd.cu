/* 
 * purpose:      just a simple example of using CUSOLVER to obtain all
 *               eigenvalues and corresponding eigenvectors
 *               for a 5 x 5 symmetric matrix
 * ref:          https://docs.nvidia.com/cuda/cusolver/index.html#eig_examples
 * compile:      nvcc chck_cusolver_syevd.cu -lcudart -lcusolver
 * result:       Eigenvalues
 *                  -11.07     -6.23      0.86      8.87     16.09
 *               Corresponding Eigenvectors
 *                   -0.30     -0.61     -0.40     -0.37      0.49
 *                   -0.51     -0.29      0.41     -0.36     -0.61
 *                   -0.08     -0.38      0.66      0.50      0.40
 *                   -0.00     -0.45     -0.46      0.62     -0.46
 *                   -0.80      0.45     -0.17      0.31      0.16
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>


int main(int argc, char **argv) 
{
   int i, j, lwork, info_gpu, *devInfo;
   double *d_A, *d_W, *d_work;
   hipsolverHandle_t cusolverH;
   hipsolverStatus_t cusolver_status;
   hipsolverEigMode_t jobz;
   hipblasFillMode_t uplo;
   hipError_t cudaStat;
   const int m = 5;
   const int lda = m;
   double W[m]; 
   double V[lda*m];
   double A[lda*m] = { 1.96, -6.49, -0.47, -7.20, -0.65, 
                      -6.49,  3.80, -6.39,  1.50, -6.34,
                      -0.47, -6.39,  4.17, -1.51,  2.67,
                      -7.20,  1.50, -1.51,  5.70,  1.80,
                      -0.65, -6.34,  2.67,  1.80, -7.10};
   jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors
   uplo = HIPBLAS_FILL_MODE_LOWER; 


   // print out initial matrix content
   printf(" Matrix to be sent into cusolverDnDsyevd\n");
   for (i = 0; i < lda; i++) {
       for (j = 0; j < m; j++) {
           printf("%10.2lf", A[(i*lda)+j]);
       }
       printf("\n");
   }


   // step 1: create cusolver/cublas handle
   cusolver_status = hipsolverDnCreate(&cusolverH);
   assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

   // step 2: copy matrices to device
   cudaStat = hipMalloc ((void**)&d_A, sizeof(double) * lda * m);
   assert(hipSuccess == cudaStat);
   cudaStat = hipMalloc ((void**)&d_W, sizeof(double) * m);
   assert(hipSuccess == cudaStat);
   cudaStat = hipMalloc ((void**)&devInfo, sizeof(int));
   assert(hipSuccess == cudaStat);
   cudaStat = hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice);
   assert(hipSuccess == cudaStat);

   // step 3: query working space of syevd
   cusolver_status = hipsolverDnDsyevd_bufferSize(cusolverH, jobz, uplo, m, d_A, lda, d_W, &lwork);
   assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
   cudaStat = hipMalloc((void**)&d_work, sizeof(double)*lwork);
   assert(hipSuccess == cudaStat);

   // step 4: compute eigenvalues/eigenvectors
   cusolver_status = hipsolverDnDsyevd(cusolverH, jobz, uplo, m, d_A, lda, d_W, d_work, lwork, devInfo);
   assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
   cudaStat = hipDeviceSynchronize();
   assert(hipSuccess == cudaStat);

   // step 5: retrieve the results from device memory
   cudaStat = hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost);
   assert(hipSuccess == cudaStat);
   cudaStat = hipMemcpy(V, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
   assert(hipSuccess == cudaStat);
   cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
   assert(hipSuccess == cudaStat);

   // step 6: print out results, ie eigenvalues and corresponding eigenvectors
   printf(" Eigenvalues\n");
   for (i = 0; i < lda; i++) {
       printf("%10.2lf", W[i]);
   }
   printf("\n");
   printf(" Corresponding Eigenvectors\n");
   for (i = 0; i < lda; i++) {
       for (j = 0; j < m; j++) {
           printf("%10.2lf", V[(j*m)+i]);
       }
       printf("\n");
   }

   // step 7: free all allocated memory
   hipFree(d_A);
   hipFree(d_W);
   hipFree(devInfo);
   hipFree(d_work);
   hipsolverDnDestroy(cusolverH);
   hipDeviceReset();


   return(0);
}
