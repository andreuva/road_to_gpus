/* 
 * purpose:      just a simple check whether a matrix, A, is composed 
 *               of eigenvectors only, in which case A^t x A = E 
 *               hence the inverse, A^-1, is simply the transpose, A^t,
 *               resulting in the unit matrix, E, by the above matrix
 *               matrix multiplication;
 *               n.b. here we want to make use of CUBLAS but check out
 *                    the feasibility of CUDA-managed unified memory
 *                    rather than the forth-and-back-copied variant 
 *                    using cudaMalloc()
 * compile:      nvcc chck_ev_v3.cu -lcublas
 * result:       unfortunately, this doesn't seem to work in a 
 *               straightforward way
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"




int main(int argc, char **argv) 
{
   int N, i, j, Adim, Bdim, Cdim;
   double alpha, beta, *A, **A2D, *B, **B2D, *C, **C2D;
   hipblasStatus_t stat;
   hipblasHandle_t handle;
   hipblasOperation_t Atype, Btype;


   // memory allocation and parameter set up
   N = 5;
   Adim = N;
   Bdim = N;
   Cdim = N;
   alpha = (double) 1;
   beta = (double) 0;
  
   hipMallocManaged(&A, N * N * sizeof(double));
   hipMallocManaged(&B, N * N * sizeof(double));
   hipMallocManaged(&C, N * N * sizeof(double));
   A2D = (double **) malloc(N * sizeof(double *));
   B2D = (double **) malloc(N * sizeof(double *));
   C2D = (double **) malloc(N * sizeof(double *));
   for (i = 0; i < N; i++) {
       A2D[i] = (double *) malloc(N * sizeof(double));
       B2D[i] = (double *) malloc(N * sizeof(double));
       C2D[i] = (double *) malloc(N * sizeof(double));
   }

   // set up matrix A2D[][] supposedly consisting of just eigenvectors
   A2D[0][0] =  0.30; A2D[0][1] = -0.61; A2D[0][2] =  0.40; A2D[0][3] =  0.37; A2D[0][4] = -0.49;
   A2D[1][0] =  0.51; A2D[1][1] = -0.29; A2D[1][2] = -0.41; A2D[1][3] =  0.36; A2D[1][4] =  0.61;
   A2D[2][0] =  0.08; A2D[2][1] = -0.38; A2D[2][2] = -0.66; A2D[2][3] = -0.50; A2D[2][4] = -0.40;
   A2D[3][0] =  0.00; A2D[3][1] = -0.45; A2D[3][2] =  0.46; A2D[3][3] = -0.62; A2D[3][4] =  0.46;
   A2D[4][0] =  0.80; A2D[4][1] =  0.45; A2D[4][2] =  0.17; A2D[4][3] = -0.31; A2D[4][4] = -0.16;

   // get the inverse of A2D[][] from simply the transpose (if really just eigenvectors)
   for (i = 0; i < N; i++) {
       for (j = 0; j < N; j++) {
           B2D[i][j] = A2D[j][i];
       }
   }

   // print out initial matrix content
   printf(" Matrix to be sent into DGEMM\n");
   for (i = 0; i < N; i++) {
       for (j = 0; j < N; j++) {
           printf("%10.2lf", A2D[i][j]); 
       }
       printf("\n");
   }      

   // copy content of A2D[][] and B2D[][] into their linear versions A[] and B[] --- column wise !
   for (i = 0; i < N; i++) {
       for (j = 0; j < N; j++) {
           A[(i*N)+j] = A2D[j][i];
           B[(i*N)+j] = B2D[j][i];
       }
   }

   // cublas: initiate the CUBLAS context
   stat = hipblasCreate(&handle);

   // cublas: set a couple of other CUBLAS parameters 
   Atype = HIPBLAS_OP_N;
   Btype = HIPBLAS_OP_N;

   // call BLAS routine DGEMM --- only pointers as arguments !
   stat = hipblasDgemm(handle, Atype, Btype, Adim, Bdim, Cdim, &alpha, 
                      &B[0],  Bdim,  &A[0], Adim, &beta, &C[0], Cdim);
   if ( stat != HIPBLAS_STATUS_SUCCESS ) {
      printf("CUBLAS error \n");
      exit(99);
   }

   // print out results, hence the unit matrix if the assumption above was correct 
   printf(" Matrix matrix product\n");
   for (i = 0; i < N; i++) {
       for (j = 0; j < N; j++) {
           C2D[i][j] = C[(j*N)+i];
           printf("%10.2lf", C2D[i][j]);
       }
       printf("\n");
   }

   // and free up allocated memory
   for (i = N-1; i >= 0; i--) {
       free(C2D[i]);
       free(B2D[i]);
       free(A2D[i]);
   }
   free(C2D);
   free(B2D);
   free(A2D);



   return(0);
}
