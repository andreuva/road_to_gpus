/* 
 purpose:      Compute the eigenvalues and eigenvectors of a matrix A
               of N*N with N=5
 compile:      nvcc eigenvalues_vectors.cu -lcusolver
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>


void printMatrixArray(int m, int n, const double *A, int lda, const char* name){
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\t", name, row+1, col+1, Areg);
        }
        printf("\n");
    }
    printf("\n");
}

void printMatrix(int m, int n, double **A, int lda, const char* name){
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row][col];
            printf("%s(%d,%d) = %f\t", name, row+1, col+1, Areg);
        }
        printf("\n");
    }
    printf("\n");
}


int main(int argc, char **argv){
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipsolverHandle_t cusolver_handle = NULL;
    hipError_t cudaStat = hipSuccess;

    hipsolverEigMode_t jobz;
    hipblasFillMode_t uplo;

    int N, i, j;
    double *A, **A_host, *Evectors, *Evalues;
    double *device_A = NULL;
    double *device_Eigenvalues = NULL;
    int *device_info = NULL;
    int info_gpu

    // memory allocation and parameter set up
    N = 5;

    A = (double *) malloc(N * N * sizeof(double));
    Evectors = (double *) malloc(N * N * sizeof(double));
    Evalues = (double *) malloc(N * sizeof(double));
    A_host = (double **) malloc(N * sizeof(double *));
    for (i = 0; i < N; i++) {
        A_host[i] = (double *) malloc(N * sizeof(double));
    }

    // set up matrix A_host[][] to compute the eigenvalues
    A_host[0][0] =  1.96; A_host[0][1] = -6.49; A_host[0][2] = -0.47; A_host[0][3] = -7.20; A_host[0][4] = -0.65;
    A_host[1][0] = -6.49; A_host[1][1] =  3.80; A_host[1][2] = -6.39; A_host[1][3] =  1.50; A_host[1][4] = -6.34;
    A_host[2][0] = -0.47; A_host[2][1] = -6.39; A_host[2][2] =  4.17; A_host[2][3] = -1.51; A_host[2][4] =  2.67;
    A_host[3][0] = -7.20; A_host[3][1] =  1.50; A_host[3][2] = -1.51; A_host[3][3] =  5.70; A_host[3][4] =  1.80;
    A_host[4][0] = -0.65; A_host[4][1] = -6.34; A_host[4][2] =  2.67; A_host[4][3] =  1.80; A_host[4][4] = -7.10;

    // copy content of A_host[][] into their linear version A[] --- column wise !
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            A[(i*N)+j] = A_host[i][j];
        }
    }

    printf("================================================")
    printMatrix(N, N, A_host, N, "A_host");
    printMatrixArray(N, N, A, N, "A");
    printf("================================================")

    jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors
    uplo = HIPBLAS_FILL_MODE_LOWER; 

    // Step 1: initiate the cusolverDn context
    cusolver_status = hipsolverDnCreate(&cusolver_handle);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    // step 2: copy matrices to device
    cudaStat = hipMalloc ((void**)&device_A, sizeof(double) * N * N);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc ((void**)&device_Eigenvalues, sizeof(double) * N);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc ((void**)&device_info, sizeof(int));
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(device_A, A, sizeof(double) * N * N, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);


    // step 5: retrieve the results from device memory
    cudaStat = hipMemcpy(Evalues, device_Eigenvalues, sizeof(double)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(Evectors, device_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(&info_gpu, device_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat);

    // step 6: print out results, ie eigenvalues and corresponding eigenvectors
    printf(" Eigenvalues\n");
    printMatrixArray(1, N, Evalues, 1, "lambda");
    printf("================================================")
    printf(" Corresponding Eigenvectors\n");
    printMatrixArray(N, N, Evectors, N, "V");
    printf("================================================")
    printf("\n");
    
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            printf("%10.2lf", Evectors[(j*m)+i]);
        }
        printf("\n");
    }
  
    // step 7: free all allocated memory and destroy context
   hipFree(device_A);
   hipFree(device_Eigenvalues);
   hipFree(device_info);
   
   hipsolverDnDestroy(cusolver_handle);
   hipDeviceReset();

    for (i = N-1; i >= 0; i--) {
        free(A_host[i]);
    }
    free(A_host);
    free(A);

    return(0);
}
